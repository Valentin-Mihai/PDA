#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include  <time.h>
#include <sys/time.h>
#define inf 99999

__global__ void funct1(int n, int k, float* x, int* qx) {

	__shared__ float dBlock[1024];
	__shared__ float QBlock[1024];
	int i = (threadIdx.x >> 5);
	int j = threadIdx.x & 31;

	int index1 = (k * 32 + i) * n + k * 32 + j;
	dBlock[threadIdx.x] = x[index1];
	QBlock[threadIdx.x] = qx[index1];
	int k1 = k * 32;

	for (int l = 0; l < 32; l++) {
		syncthreads();
		float temp2 = dBlock[(i << 5) + l] + dBlock[(l << 5) + j];
		if (dBlock[threadIdx.x] > temp2) {
			dBlock[threadIdx.x] = temp2;
			QBlock[threadIdx.x] = l + k1;
		}
	}
	x[index1] = dBlock[threadIdx.x];
	qx[index1] = QBlock[threadIdx.x];
}

__global__ void funct2(int n, int k, float* x, int* qx) {
	if (blockIdx.y == 0) {

		int i = (threadIdx.x >> 5);
		int j = threadIdx.x & 31;
		int k1 = k * 32;
		__shared__ float dBlock[1024];
		__shared__ float QcBlock[1024];
		__shared__ float cBlock[1024];
		dBlock[threadIdx.x] = x[(k1 + i) * n + k1 + j];
		int add = 0;

		if (blockIdx.x >= k) { //jumping over central block
			add = 1;
		}

		int index1 = (k1 + i) * n + (blockIdx.x + add) * 32 + j;
		cBlock[threadIdx.x] = x[index1];
		QcBlock[threadIdx.x] = qx[index1];

		for (int l = 0; l < 32; l++) {
			syncthreads();
			float temp2 = dBlock[i * 32 + l] + cBlock[l * 32 + j];
			if (cBlock[threadIdx.x] > temp2) {
				cBlock[threadIdx.x] = temp2;
				QcBlock[threadIdx.x] = l + k1;
			}
		}
		x[index1] = cBlock[threadIdx.x];
		qx[index1] = QcBlock[threadIdx.x];

	}
	else {

		int i = (threadIdx.x >> 5);
		int j = threadIdx.x & 31;
		int k1 = k * 32;
		__shared__ float dBlock[1024];
		__shared__ float QcBlock[1024];
		__shared__ float cBlock[1024];
		dBlock[threadIdx.x] = x[(k1 + i) * n + k1 + j];
		int add = 0;

		if (blockIdx.x >= k) { //jumping over central block        
			add = 1;
		}

		int index1 = ((blockIdx.x + add) * 32 + i) * n + k1 + j;
		cBlock[threadIdx.x] = x[index1];
		QcBlock[threadIdx.x] = qx[index1];

		for (int l = 0; l < 32; l++) {
			syncthreads();
			float temp2 = cBlock[i * 32 + l] + dBlock[l * 32 + j];

			if (cBlock[threadIdx.x] > temp2) {
				cBlock[threadIdx.x] = temp2;

				QcBlock[threadIdx.x] = l + k1;
			}
		}
		x[index1] = cBlock[threadIdx.x];
		qx[index1] = QcBlock[threadIdx.x];
	}
}

__global__ void funct3(int n, int k, float* x, int* qx) {
	int i = (threadIdx.x >> 5);
	int j = threadIdx.x & 31;
	int k1 = k * 32;
	int addx = 0;
	int addy = 0;

	__shared__ float dyBlock[1024];
	__shared__ float dxBlock[1024];
	__shared__ float QcBlock[1024];
	__shared__ float cBlock[1024];

	if (blockIdx.x >= k) {
		addx = 1;

	}
	if (blockIdx.y >= k) {
		addy = 1;

	}

	dxBlock[threadIdx.x] = x[((k << 5) + i) * n + ((blockIdx.y + addy) << 5) + j];
	dyBlock[threadIdx.x] = x[(((blockIdx.x + addx) << 5) + i) * n + (k << 5) + j];
	int index1 = (((blockIdx.x + addx) << 5) + i) * n + ((blockIdx.y + addy) << 5) + j;
	cBlock[threadIdx.x] = x[index1];
	QcBlock[threadIdx.x] = qx[index1];

	for (int l = 0; l < 32; l++) {
		syncthreads();
		float temp2 = dyBlock[i * 32 + l] + dxBlock[l * 32 + j];
		if (cBlock[threadIdx.x] > temp2) {
			cBlock[threadIdx.x] = temp2;
			QcBlock[threadIdx.x] = l + k1;
		}
	}
	x[index1] = cBlock[threadIdx.x];
	qx[index1] = QcBlock[threadIdx.x];
}


int main(int argc, char **argv) {

	struct timeval first, second, lapsed, third, fourth, lapsed2;
	struct timezone tzp, tzp2;
	float *host_A, *host_D;
	int *host_Q;
	float *dev_x;
	int *dev_qx;
	float *A;
	int *Q;
	float *D;

	int i, j;
	int k = 0;
	float tolerance = 0.001;
	int n = atoi(argv[1]);

	printf("\n");
	printf("RUNNING WITH %d VERTICES \n", n);
	printf("\n");
	hipMalloc(&dev_x, n * n * sizeof(float));
	hipMalloc(&dev_qx, n * n * sizeof(float));

	//CPU arrays
	A = (float *)malloc(n * n * sizeof(float));
	D = (float *)malloc(n * n * sizeof(float));
	Q = (int *)malloc(n * n * sizeof(int));

	//GPU arrays
	host_A = (float *)malloc(n * n * sizeof(float));
	host_D = (float *)malloc(n * n * sizeof(float));
	host_Q = (int *)malloc(n * n * sizeof(int));

	srand(time(NULL));

	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			Q[i * n + j] = -1;
		}
	}
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			if (i == j) {
				A[i * n + j] = 0;
			}
			else {
				A[i * n + j] = 1200 * (float)rand() / RAND_MAX + 1;
			
				if (A[i * n + j] > 1000) {
					A[i * n + j] = inf;
					Q[i * n + j] = -2;
				}
			}
		}
	}
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			D[i * n + j] = A[i * n + j];
		}

	}
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			host_A[i * n + j] = A[i * n + j];
		}

	}
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			host_Q[i * n + j] = Q[i * n + j];
		}

	}

	printf("GPU running... \n");
	gettimeofday(&third, &tzp2);
	////////////////////////////First Mem Copy////////////////////
	gettimeofday(&first, &tzp);
	hipMemcpy(dev_x, host_A, n * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_qx, host_Q, n * n * sizeof(int), hipMemcpyHostToDevice);
	gettimeofday(&second, &tzp);
	if (first.tv_usec > second.tv_usec) {
		second.tv_usec += 1000000;
		second.tv_sec--;
	}
	lapsed.tv_usec = second.tv_usec - first.tv_usec;
	lapsed.tv_sec = second.tv_sec - first.tv_sec;
	printf("First Transfer CPU to GPU  Time elapsed: %lu,%06lu s\n", lapsed.tv_sec, lapsed.tv_usec);
	printf("\n");
	////////////////////////////////////////////////////GPU Calculation////////////////////////////////
	gettimeofday(&first, &tzp);
	dim3 bk2(n / 32 - 1, 2);
	dim3 bk3(n / 32 - 1, n / 32 - 1);
	int gputhreads = 1024;
	for (k = 0; k < n / 32; k++) {
		funct1 << <1, gputhreads >> >(n, k, dev_x, dev_qx);
		funct2 << <bk2, gputhreads >> >(n, k, dev_x, dev_qx);
		funct3 << <bk3, gputhreads >> >(n, k, dev_x, dev_qx);
	}
	hipDeviceSynchronize();
	gettimeofday(&second, &tzp);
	if (first.tv_usec > second.tv_usec) {
		second.tv_usec += 1000000;
		second.tv_sec--;
	}

	lapsed.tv_usec = second.tv_usec - first.tv_usec;
	lapsed.tv_sec = second.tv_sec - first.tv_sec;
	printf("GPU Calculation Time elapsed: %lu,%06lu s\n", lapsed.tv_sec, lapsed.tv_usec);
	printf("\n");
	//////////////////////////////////////////////////////////////////////////Second Mem Copy////////////////////
	gettimeofday(&first, &tzp);
	hipMemcpy(host_D, dev_x, n * n * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(host_Q, dev_qx, n * n * sizeof(int), hipMemcpyDeviceToHost);
	gettimeofday(&second, &tzp);
	if (first.tv_usec > second.tv_usec) {
		second.tv_usec += 1000000;
		second.tv_sec--;
	}
	lapsed.tv_usec = second.tv_usec - first.tv_usec;
	lapsed.tv_sec = second.tv_sec - first.tv_sec;
	printf("Second Transfer GPU to CPU  Time elapsed: %lu,%06lu s\n", lapsed.tv_sec, lapsed.tv_usec);
	printf("\n");
	//////////////////////////////////////////////////////////////////////

	gettimeofday(&fourth, &tzp2); //total time
	if (third.tv_usec > fourth.tv_usec) {
		fourth.tv_usec += 1000000;
		fourth.tv_sec--;
	}
	lapsed2.tv_usec = fourth.tv_usec - third.tv_usec;
	lapsed2.tv_sec = fourth.tv_sec - third.tv_sec;
	printf("TOTAL GPU + TRANSFERS  Time elapsed: %lu,%06lu s\n", lapsed2.tv_sec, lapsed2.tv_usec);
	printf("\n");
	//////////////////////////////////////////////////////////////
	//CPU RUN 

	printf("\n");
	printf("\n");
	printf(" Now running on CPU... \n");
	printf("\n");
	gettimeofday(&first, &tzp);
	for (k = 0; k < n; k++) {
		for (i = 0; i < n; i++) {
			for (j = 0; j < n; j++) {

				if ((D[i * n + k] + D[k * n + j]) < D[i * n + j]) {
					D[i * n + j] = D[i * n + k] + D[k * n + j];
					Q[i * n + j] = k;
				}
				if (D[i * n + j] == inf) {
					//Q[i*n+j]=-2;
				}
			}
		}
	}
	/////////////////////////////////////////////////////////////////
	gettimeofday(&second, &tzp);
	if (first.tv_usec > second.tv_usec) {
		second.tv_usec += 1000000;
		second.tv_sec--;
	}
	lapsed.tv_usec = second.tv_usec - first.tv_usec;
	lapsed.tv_sec = second.tv_sec - first.tv_sec;
	printf("CPU Time elapsed: %lu,%06lu s\n", lapsed.tv_sec, lapsed.tv_usec);
	/////////////////////////////////////////////////////
	printf(" \n");
	printf(" \n");
	/////////////FROM HERE AND UNDER ARE VALIDATION RUNS

	printf("VALIDATING THAT D array from CPU and host_D array from GPU match... \n");
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			if (abs(D[i * n + j] - host_D[i * n + j]) > tolerance) {

				printf("ERROR MISMATCH in array D i %d j %d CPU SAYS %f and GPU SAYS %f \n", i, j, D[i * n + j], host_D[i * n + j]);
			}
		}
	}
	printf("OK \n");

	printf("ALL OK WE ARE DONE \n");
	return 0;
}
